#include "hip/hip_runtime.h"
#pragma once 


#include "gauss_newton_solver.h"
#include "util.h"
#include "device_util.h"
#include "device_array.h"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"

__global__ void cuComputeJacobianSparseFeatures(
	//shared memory
	const int nFeatures, const int imageWidth, const int imageHeight,
	const int nShapeCoeffs, const int nExpressionCoeffs, const int nAlbedoCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal, const int nAlbedoCoeffsTotal,
	const float sqrt_wreg,

	uchar* image,

	glm::mat4 face_pose, glm::mat3 drx, glm::mat3 dry, glm::mat3 drz, glm::mat4 projection, Eigen::Matrix3f jacobian_local,

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features,

	float* p_shape_basis,
	float* p_expression_basis,
	float* p_albedo_basis,

	float* p_coefficients_shape,
	float* p_coefficients_expression,
	float* p_coefficients_albedo,

	hipTextureObject_t rgb,
	hipTextureObject_t barycentrics,
	hipTextureObject_t vertex_ids,

	//device memory output
	float* p_jacobian, float* p_residuals)
{
	int index = util::getThreadIndex1D();
	int stride = blockDim.x * gridDim.x;

	const int nFaceCoeffs = nShapeCoeffs + nExpressionCoeffs + nAlbedoCoeffs;
	const int nPixels = imageWidth * imageHeight;
	const int n = nFeatures + nPixels + nFaceCoeffs;

	Eigen::Map<Eigen::MatrixXf> jacobian(p_jacobian, nResiduals, nUnknowns);
	Eigen::Map<Eigen::VectorXf> residuals(p_residuals, nResiduals);

	Eigen::Map<Eigen::MatrixXf> shape_basis(p_shape_basis, nVerticesTimes3, nShapeCoeffsTotal);
	Eigen::Map<Eigen::MatrixXf> expression_basis(p_expression_basis, nVerticesTimes3, nExpressionCoeffsTotal);
	Eigen::Map<Eigen::MatrixXf> albedo_basis(p_albedo_basis, nVerticesTimes3, nAlbedoCoeffsTotal);

	int offset_rows = nFeatures * 2 + nPixels * 3;
	int offset_cols = 7;

	for (int i = index; i < n; i += stride)
	{
		// Regularization terms

		if (i >= nResiduals - nFaceCoeffs)
		{
			const int shape_expression = nShapeCoeffs + nExpressionCoeffs;
			const int current_index = i - nResiduals - nFaceCoeffs;
			// Range of offset for each coefficient
			// 0 -> nShapeCoeffs -> (nShapeCoeffs + nExpressionCoeffs)
			const int shift = current_index >= nShapeCoeffs ?
				current_index >= shape_expression ? shape_expression : nShapeCoeffs : 0;

			offset_rows += shift;
			offset_cols += shift;

			const int relative_index = current_index - shift;

			// Depending which shift is used the proper coefficient is selected
			const float coefficient = shift == 0 ? p_coefficients_shape[relative_index] : shift == nShapeCoeffs ?
				p_coefficients_expression[relative_index] : p_coefficients_albedo[relative_index];

			jacobian(offset_rows + relative_index, offset_cols + relative_index) = sqrt_wreg;
			residuals(offset_rows + relative_index) = coefficient * sqrt_wreg;

			return;
		}

		// Dense terms

		if (i >= nFeatures)
		{
			int idx = i - nFeatures;
			int xp = idx % imageWidth;
			int yp = idx / imageWidth;
			idx *= 3;

			int ygl = imageHeight - 1 - yp; // "height - 1 - index.y" OpenGL uses left-bottom corner as texture origin.
			float4 face_rgb_sampled = tex2D<float4>(rgb, xp, ygl);

			if (face_rgb_sampled.w < 1.0f) return; // pixel is not covered by face

			float4 bary_sampled = tex2D<float4>(barycentrics, xp, ygl);
			int4 verts_s = tex2D<int4>(vertex_ids, xp, ygl);
			Eigen::Map<Eigen::Vector3f> face_rgb(reinterpret_cast<float*>(&face_rgb_sampled));
			Eigen::Vector3f frame_rgb;

			frame_rgb.x() = image[idx] / 255.0f;
			frame_rgb.y() = image[idx + 1] / 255.0f;
			frame_rgb.z() = image[idx + 2] / 255.0f;

			Eigen::Vector3f residual = face_rgb - frame_rgb;

			residuals.block(i * 3, 0, 3, 1) = residual;

			// Albedo

			auto light = bary_sampled.w;

			auto A = light * bary_sampled.x * albedo_basis.block(3 * verts_s.x, 0, 3, nAlbedoCoeffs);
			auto B = light * bary_sampled.y * albedo_basis.block(3 * verts_s.y, 0, 3, nAlbedoCoeffs);
			auto C = light * bary_sampled.z * albedo_basis.block(3 * verts_s.z, 0, 3, nAlbedoCoeffs);

			jacobian.block(i * 3, 7 + nShapeCoeffs + nExpressionCoeffs, 3, nAlbedoCoeffs) = A + B + C;

			// Shape and expression

			jacobian.block(i * 3, 7, 3, nShapeCoeffs) = Eigen::MatrixXf::Zero(3, nShapeCoeffs);
			jacobian.block(i * 3, 7 + nShapeCoeffs, 3, nExpressionCoeffs) = Eigen::MatrixXf::Zero(3, nExpressionCoeffs);

			return;
		}

		// Sparse terms

		Eigen::Matrix<float, 2, 3> jacobian_proj = Eigen::MatrixXf::Zero(2, 3);

		Eigen::Matrix<float, 3, 3> jacobian_world = Eigen::MatrixXf::Zero(3, 3);
		jacobian_world(1, 1) = projection[1][1];
		jacobian_world(2, 2) = -1.0f;

		Eigen::Matrix<float, 3, 1> jacobian_intrinsics = Eigen::MatrixXf::Zero(3, 1);

		Eigen::Matrix<float, 3, 6> jacobian_pose = Eigen::MatrixXf::Zero(3, 6);
		jacobian_pose(0, 3) = 1.0f;
		jacobian_pose(1, 4) = 1.0f;
		jacobian_pose(2, 5) = 1.0f;

		auto vertex_id = prior_local_ids[i];
		auto local_coord = current_face[vertex_id];

		auto world_coord = face_pose * glm::vec4(local_coord, 1.0f);
		auto proj_coord = projection * world_coord;
		auto uv = glm::vec2(proj_coord.x, proj_coord.y) / proj_coord.w;

		//Residual
		auto residual = uv - sparse_features[i];

		residuals(i * 2) = residual.x;
		residuals(i * 2 + 1) = residual.y;

		//Jacobian for homogenization (AKA division by w)
		auto one_over_wp = 1.0f / proj_coord.w;
		jacobian_proj(0, 0) = one_over_wp;
		jacobian_proj(0, 2) = -proj_coord.x * one_over_wp * one_over_wp;

		jacobian_proj(1, 1) = one_over_wp;
		jacobian_proj(1, 2) = -proj_coord.y * one_over_wp * one_over_wp;

		//Jacobian for projection
		jacobian_world(0, 0) = projection[0][0];

		//Jacobian for intrinsics
		jacobian_intrinsics(0, 0) = world_coord.x;
		jacobian.block<2, 1>(i * 2, 0) = jacobian_proj * jacobian_intrinsics;

		//Derivative of world coordinates with respect to rotation coefficients
		auto dx = drx * local_coord;
		auto dy = dry * local_coord;
		auto dz = drz * local_coord;

		jacobian_pose(0, 0) = dx[0];
		jacobian_pose(1, 0) = dx[1];
		jacobian_pose(2, 0) = dx[2];
		jacobian_pose(0, 1) = dy[0];
		jacobian_pose(1, 1) = dy[1];
		jacobian_pose(2, 1) = dy[2];
		jacobian_pose(0, 2) = dz[0];
		jacobian_pose(1, 2) = dz[1];
		jacobian_pose(2, 2) = dz[2];

		auto jacobian_proj_world = jacobian_proj * jacobian_world;
		jacobian.block<2, 6>(i * 2, 1) = jacobian_proj_world * jacobian_pose;

		//Derivative of world coordinates with respect to local coordinates.
		//This is basically the rotation matrix.
		auto jacobian_proj_world_local = jacobian_proj_world * jacobian_local;

		//Derivative of local coordinates with respect to shape and expression parameters
		//This is basically the corresponding (to unique vertices we have chosen) rows of basis matrices.
		auto jacobian_shape = jacobian_proj_world_local * shape_basis.block(3 * vertex_id, 0, 3, nShapeCoeffs);
		jacobian.block(i * 2, 7, 2, nShapeCoeffs) = jacobian_shape;

		auto jacobian_expression = jacobian_proj_world_local * expression_basis.block(3 * vertex_id, 0, 3, nExpressionCoeffs);
		jacobian.block(i * 2, 7 + nShapeCoeffs, 2, nExpressionCoeffs) = jacobian_expression;
	}
}

void GaussNewtonSolver::computeJacobianSparseFeatures(
	//shared memory
	const int nFeatures, const int imageWidth, const int imageHeight,
	const int nShapeCoeffs, const int nExpressionCoeffs, const int nAlbedoCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal, const int nAlbedoCoeffsTotal,
	const float regularizationWeight,

	uchar* image,

	const glm::mat4& face_pose, const glm::mat3& drx, const glm::mat3& dry, const glm::mat3& drz, const glm::mat4& projection, const Eigen::Matrix3f& jacobian_local,

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features,

	float* p_shape_basis,
	float* p_expression_basis,
	float* p_albedo_basis,

	float* p_coefficients_shape,
	float* p_coefficients_expression,
	float* p_coefficients_albedo,

	//device memory output
	float* p_jacobian, float* p_residuals
) const
{
	const int nPixels = imageWidth * imageHeight;
	const int n = nFeatures + nPixels + m_params.num_shape_coefficients + m_params.num_expression_coefficients + m_params.num_albedo_coefficients;

	const int threads = 256;
	const int block = (n + threads - 1) / threads;

	cuComputeJacobianSparseFeatures << <block, threads >> > (
		//shared memory
		nFeatures, imageWidth, imageHeight,
		nShapeCoeffs, nExpressionCoeffs, nAlbedoCoeffs,
		nUnknowns, nResiduals,
		nVerticesTimes3, nShapeCoeffsTotal, nExpressionCoeffsTotal, nAlbedoCoeffsTotal,
		glm::sqrt(regularizationWeight),

		image,

		face_pose, drx, dry, drz, projection, jacobian_local,

		//device memory input
		prior_local_ids, current_face, sparse_features,

		p_shape_basis,
		p_expression_basis,
		p_albedo_basis,

		p_coefficients_shape,
		p_coefficients_expression,
		p_coefficients_albedo,

		m_texture_rgb,
		m_texture_barycentrics,
		m_texture_vertex_ids,

		//device memory output
		p_jacobian, p_residuals);

	hipDeviceSynchronize();
}

__global__ void cuComputeJacobiPreconditioner(const int nUnknowns, const int nResiduals, float* p_jacobian, float* p_preconditioner)
{
	extern __shared__ float temp[];

	int col = blockIdx.x;
	int row = threadIdx.x;
	float v = p_jacobian[col * nResiduals + row];
	temp[row] = v * v;

	__syncthreads();

	if (threadIdx.x == 0)
	{
		float sum = 0;
		for (int i = 0; i < nResiduals; i++)
		{
			sum += temp[i];
		}
		p_preconditioner[col] = 1.0f / (fmaxf(2.0f*sum, 1e-8f));
	}

}

__global__ void cuElementwiseMultiplication(float* v1, float* v2, float* out)
{
	int i = util::getThreadIndex1D();
	out[i] = v1[i] * v2[i];
}

void GaussNewtonSolver::computeJacobiPreconditioner(const int nUnknowns, const int nResiduals, float* p_jacobian, float* p_preconditioner)
{
	//TODO: split this up into proper blocks, once we have more that 1024 resiudals 
	cuComputeJacobiPreconditioner << <nUnknowns, nResiduals, sizeof(float)*nResiduals >> > (nUnknowns, nResiduals, p_jacobian, p_preconditioner);
	hipDeviceSynchronize();
}

void GaussNewtonSolver::elementwiseMultiplication(const int nElements, float* v1, float* v2, float* out)
{
	cuElementwiseMultiplication << <1, nElements >> > (v1, v2, out);
	hipDeviceSynchronize();
}