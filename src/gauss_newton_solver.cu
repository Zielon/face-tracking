#include "hip/hip_runtime.h"
#pragma once 


#include "gauss_newton_solver.h"
#include "util.h"
#include "device_util.h"
#include "device_array.h"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
//texture<float4, hipTextureType2D, hipReadModeElementType> tex_rgb;
//texture<float4, hipTextureType2D, hipReadModeElementType> tex_barycentrics;
//texture<int4, hipTextureType2D, hipReadModeElementType> tex_vertex_ids;

__global__ void cuComputeJacobianSparseFeatures( 
	//shared memory
	const int nFeatures,
	const int nShapeCoeffs, const int nExpressionCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal,
	const float regularizationWeight,

	glm::mat4 face_pose, glm::mat3 drx, glm::mat3 dry, glm::mat3 drz, glm::mat4 projection, Eigen::Matrix3f jacobian_local, 

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features, 
	float* p_shape_basis,  float* p_expression_basis, float* p_coefficients_shape, float* p_coefficients_expression,

	//device memory output
	float* p_jacobian, float* p_residuals)
{
	int i = util::getThreadIndex1D(); 

	Eigen::Map<Eigen::MatrixXf> jacobian(p_jacobian, nResiduals, nUnknowns);
	Eigen::Map<Eigen::VectorXf> residuals(p_residuals, nResiduals);

	int offset_rows = nFeatures * 2;
	int offset_cols = 7;

	// Regularization terms
	if (i >= nFeatures)
	{
		const int current_index = i - nFeatures;
		const int shift = current_index >= nShapeCoeffs ? nShapeCoeffs : 0;

		offset_rows += shift;
		offset_cols += shift;

		const int relative_index = current_index - shift;

		const float coefficient = shift > 0 ? p_coefficients_expression[relative_index] : p_coefficients_shape[relative_index];

		auto sqrt_wreg = glm::sqrt(regularizationWeight);
		jacobian(offset_rows + relative_index, offset_cols + relative_index) = sqrt_wreg;
		residuals(offset_rows + relative_index) = coefficient * sqrt_wreg;

		return;
	}

	Eigen::Map<Eigen::MatrixXf> shape_basis(p_shape_basis, nVerticesTimes3, nShapeCoeffsTotal);
	Eigen::Map<Eigen::MatrixXf> expression_basis(p_expression_basis, nVerticesTimes3, nExpressionCoeffsTotal);

	Eigen::Matrix<float, 2, 3> jacobian_proj = Eigen::MatrixXf::Zero(2, 3);

	Eigen::Matrix<float, 3, 3> jacobian_world = Eigen::MatrixXf::Zero(3, 3);
	jacobian_world(1, 1) = projection[1][1];
	jacobian_world(2, 2) = -1.0f;

	Eigen::Matrix<float, 3, 1> jacobian_intrinsics = Eigen::MatrixXf::Zero(3, 1);

	Eigen::Matrix<float, 3, 6> jacobian_pose = Eigen::MatrixXf::Zero(3, 6);
	jacobian_pose(0, 3) = 1.0f;
	jacobian_pose(1, 4) = 1.0f;
	jacobian_pose(2, 5) = 1.0f;

	auto vertex_id = prior_local_ids[i];
	auto local_coord = current_face[vertex_id];

	auto world_coord = face_pose * glm::vec4(local_coord, 1.0f);
	auto proj_coord = projection * world_coord;
	auto uv = glm::vec2(proj_coord.x, proj_coord.y) / proj_coord.w;

	//Residual
	auto residual = uv - sparse_features[i];

	residuals(i * 2) = residual.x;
	residuals(i * 2 + 1) = residual.y;

	//Jacobian for homogenization (AKA division by w)
	auto one_over_wp = 1.0f / proj_coord.w;
	jacobian_proj(0, 0) = one_over_wp;
	jacobian_proj(0, 2) = -proj_coord.x * one_over_wp * one_over_wp;

	jacobian_proj(1, 1) = one_over_wp;
	jacobian_proj(1, 2) = -proj_coord.y * one_over_wp * one_over_wp;

	//Jacobian for projection
	jacobian_world(0, 0) = projection[0][0];

	//Jacobian for intrinsics
	jacobian_intrinsics(0, 0) = world_coord.x;
	jacobian.block<2, 1>(i * 2, 0) = jacobian_proj * jacobian_intrinsics;

	//Derivative of world coordinates with respect to rotation coefficients
	auto dx = drx * local_coord;
	auto dy = dry * local_coord;
	auto dz = drz * local_coord;

	jacobian_pose(0, 0) = dx[0];
	jacobian_pose(1, 0) = dx[1];
	jacobian_pose(2, 0) = dx[2];
	jacobian_pose(0, 1) = dy[0];
	jacobian_pose(1, 1) = dy[1];
	jacobian_pose(2, 1) = dy[2];
	jacobian_pose(0, 2) = dz[0];
	jacobian_pose(1, 2) = dz[1];
	jacobian_pose(2, 2) = dz[2];

	auto jacobian_proj_world = jacobian_proj * jacobian_world;
	jacobian.block<2, 6>(i * 2, 1) = jacobian_proj_world * jacobian_pose;

	//Derivative of world coordinates with respect to local coordinates.
	//This is basically the rotation matrix.
	auto jacobian_proj_world_local = jacobian_proj_world * jacobian_local;

	//Derivative of local coordinates with respect to shape and expression parameters
	//This is basically the corresponding (to unique vertices we have chosen) rows of basis matrices.
	auto jacobian_shape = jacobian_proj_world_local * shape_basis.block(3 * vertex_id, 0, 3, nShapeCoeffs);
	jacobian.block(i * 2, 7, 2, nShapeCoeffs) = jacobian_shape;

	auto jacobian_expression = jacobian_proj_world_local * expression_basis.block(3 * vertex_id, 0, 3, nExpressionCoeffs);
	jacobian.block(i * 2, 7 + nShapeCoeffs, 2, nExpressionCoeffs) = jacobian_expression;
}

void GaussNewtonSolver::computeJacobianSparseFeatures(
	//shared memory
	const int nFeatures,
	const int nShapeCoeffs, const int nExpressionCoeffs,
	const int nUnknowns, const int nResiduals,
	const int nVerticesTimes3, const int nShapeCoeffsTotal, const int nExpressionCoeffsTotal,
	const float regularizationWeight,

	const glm::mat4& face_pose, const glm::mat3& drx, const glm::mat3& dry, const glm::mat3& drz, const glm::mat4& projection, const Eigen::Matrix3f& jacobian_local,

	//device memory input
	int* prior_local_ids, glm::vec3* current_face, glm::vec2* sparse_features,
	float* p_shape_basis, float* p_expression_basis, float* p_coefficients_shape, float* p_coefficients_expression,

	//device memory output
	float* p_jacobian, float* p_residuals
) const 
{
	const int threads = nFeatures + m_params.num_shape_coefficients + m_params.num_expression_coefficients;

	cuComputeJacobianSparseFeatures<<<1, threads>>> (
		//shared memory
		nFeatures,
		nShapeCoeffs, nExpressionCoeffs,
		nUnknowns, nResiduals,
		nVerticesTimes3, nShapeCoeffsTotal, nExpressionCoeffsTotal,
		regularizationWeight,

		face_pose, drx, dry, drz, projection, jacobian_local,

		//device memory input
		prior_local_ids, current_face, sparse_features,
		p_shape_basis, p_expression_basis, p_coefficients_shape, p_coefficients_expression,

		//device memory output
		p_jacobian, p_residuals
	);

	hipDeviceSynchronize();
}

__global__ void cuComputeJacobiPreconditioner(const int nUnknowns, const int nResiduals, float* p_jacobian, float* p_preconditioner)
{
	extern __shared__ float temp[];

	int col = blockIdx.x;
	int row = threadIdx.x;
	float v = p_jacobian[col * nResiduals + row];
	temp[row] = v * v;

	__syncthreads();

	if (threadIdx.x == 0)
	{
		float sum = 0;
		for (int i = 0; i < nResiduals; i++)
		{
			sum += temp[i];
		}
		p_preconditioner[col] = 1.0f / (fmaxf(2.0f*sum, 1e-8f));
	}

}

__global__ void cuElementwiseMultiplication(float* v1, float* v2, float* out)
{
	int i = util::getThreadIndex1D();
	out[i] = v1[i] * v2[i];
}

__global__ void cuSampleTextureToVector(hipTextureObject_t tex, float4* memory, int H, int W)
{
	uint i = util::getThreadIndex1D();
	if (i < H*W)
	{
		uint x = i / W; 
		uint y = i - x * W;
		memory[i] = tex2D<float4>(tex, x, y); 
	}

}


void GaussNewtonSolver::computeJacobiPreconditioner(const int nUnknowns, const int nResiduals, float* p_jacobian, float* p_preconditioner)
{
	//TODO: split this up into proper blocks, once we have more that 1024 resiudals 
	cuComputeJacobiPreconditioner<<<nUnknowns, nResiduals, sizeof(float)*nResiduals>>>(nUnknowns, nResiduals, p_jacobian, p_preconditioner);
	hipDeviceSynchronize();
}

void GaussNewtonSolver::elementwiseMultiplication(const int nElements, float* v1, float* v2, float* out)
{
	cuElementwiseMultiplication<<<1, nElements>>>(v1, v2, out);
	hipDeviceSynchronize();
}

void GaussNewtonSolver::mapRenderTargets(Face& face)
{
	if (face.m_graphics_settings.mapped_to_cuda)
	{
		std::cout << "map called, while rts already mapped" << std::endl;
		return;
	}
	hipGraphicsResource* ressources[] = { face.m_rt_rgb_cuda_ressource, face.m_rt_barycentrics_cuda_ressource, face.m_rt_vertex_id_cuda_ressource };
	CHECK_CUDA_ERROR(hipGraphicsMapResources(3, ressources, 0));

	//will this leak? 
	hipArray* arr_rgb;
	hipArray* arr_bary;
	hipArray* arr_vert;
	hipChannelFormatDesc cfd = hipCreateChannelDesc<float4>();
	hipTextureObject_t m_tex_rgb = 0;
	hipTextureObject_t m_tex_barycentrics = 0;
	hipTextureObject_t m_tex_vertex_ids = 0;

	const textureReference* ref_tex_rgb;
	hipGetTextureReference(&ref_tex_rgb, HIP_SYMBOL(&m_tex_rgb));

	//ref_tex_rgb->normalized = 0;
	//ref_tex_rgb->filterMode = hipFilterModePoint;
	CHECK_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&arr_rgb, face.m_rt_rgb_cuda_ressource, 0, 0));
	CHECK_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&arr_bary, face.m_rt_barycentrics_cuda_ressource, 0, 0));
	CHECK_CUDA_ERROR(hipGraphicsSubResourceGetMappedArray(&arr_vert, face.m_rt_vertex_id_cuda_ressource, 0, 0));

	hipTextureDesc desc;
	memset(&desc, 0, sizeof(desc)); 
	desc.filterMode = hipFilterModePoint;
	desc.addressMode[0] = desc.addressMode[1] = desc.addressMode[2] = hipAddressModeClamp; 
	desc.normalizedCoords = false; 
	desc.readMode = hipReadModeElementType; 

	hipResourceDesc res_desc; 
	res_desc.resType = hipResourceTypeArray; 
	res_desc.res.array.array = arr_rgb; 

	hipResourceViewDesc res_view_desc;
	memset(&res_view_desc, 0, sizeof(res_view_desc));
	res_view_desc.width = face.m_graphics_settings.screen_width; 
	res_view_desc.height = face.m_graphics_settings.screen_height; 
	res_view_desc.format = hipResViewFormatFloat4; 
	

	CHECK_CUDA_ERROR(hipCreateTextureObject(&m_tex_rgb, &res_desc, &desc, 0));
	CHECK_CUDA_ERROR(hipCreateTextureObject(&m_tex_barycentrics, &res_desc, &desc, 0));
	res_view_desc.format = hipResViewFormatSignedInt4;

	CHECK_CUDA_ERROR(hipCreateTextureObject(&m_tex_vertex_ids, &res_desc, &desc, 0));

	//CHECK_CUDA_ERROR(hipBindTextureToArray(ref_tex_rgb, arr_rgb, &cfd));
	//CHECK_CUDA_ERROR(cudaBindSurfaceToArray(&surf_rgb, arr_rgb, &cfd)); 
	
	//CHECK_CUDA_ERROR(hipBindTextureToArray(&tex_barycentrics, arr_bary, &cfd));

	//cfd.f = hipChannelFormatKindSigned;
	//res_view_desc.format = hipResViewFormatSignedInt4;

	//CHECK_CUDA_ERROR(hipBindTextureToArray(&tex_vertex_ids, arr_vert, &cfd));
	face.m_graphics_settings.mapped_to_cuda = true;


	util::DeviceArray<float4> tmp(face.m_graphics_settings.screen_height*face.m_graphics_settings.screen_width) ;

	int blocks = face.m_graphics_settings.screen_height * face.m_graphics_settings.screen_width / 256 +1;

	cuSampleTextureToVector<<<blocks, 256 >>>(m_tex_rgb, tmp.getPtr(), face.m_graphics_settings.screen_height, face.m_graphics_settings.screen_width);

	std::vector<float4> v(face.m_graphics_settings.screen_height * face.m_graphics_settings.screen_width);
	util::copy(v, tmp, face.m_graphics_settings.screen_height*face.m_graphics_settings.screen_width);

	float s = 0;
	cv::Mat o = cv::Mat4f(face.m_graphics_settings.screen_width, face.m_graphics_settings.screen_height);
	cv::Mat ox = cv::Mat3b(face.m_graphics_settings.screen_width, face.m_graphics_settings.screen_height);

	for (int y = 0; y < face.m_graphics_settings.screen_height; ++y)
	{
		for (int x = 0; x < face.m_graphics_settings.screen_width; ++x)
		{
			float4 f = v[y*face.m_graphics_settings.screen_width + x];
			o.at<cv::Vec4f>(y,x) = cv::Vec4f((float*)&f);
			ox.at<cv::Vec3b>(y, x)[0] = f.x * 255;
			ox.at<cv::Vec3b>(y, x)[1] = f.y * 255;
			ox.at<cv::Vec3b>(y, x)[2] = f.z * 255;

		}
	}
	cv::Mat gdmmt; 
	
	cv::cvtColor(ox, gdmmt, cv::COLOR_RGB2BGR);
	cv::imshow("test", o);
	cv::waitKey(0); 
}

void GaussNewtonSolver::unmapRenderTargets(Face& face)
{
	if (!face.m_graphics_settings.mapped_to_cuda)
	{
		std::cout << "unmap called, while rts already unmapped" << std::endl;
		return;
	}

	//hipDestroyTextureObject(m_tex_rgb);
	//hipDestroyTextureObject(m_tex_barycentrics);
	//hipDestroyTextureObject(m_tex_vertex_ids);

	hipGraphicsResource* ressources[] = { face.m_rt_rgb_cuda_ressource, face.m_rt_barycentrics_cuda_ressource, face.m_rt_vertex_id_cuda_ressource };
	CHECK_CUDA_ERROR(hipGraphicsUnmapResources(3, ressources, 0));

	face.m_graphics_settings.mapped_to_cuda = false;
}